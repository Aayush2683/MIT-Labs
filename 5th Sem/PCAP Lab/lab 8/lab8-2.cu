//Write a program in CUDA to read MXN matrix A and replace 1 st row of this matrix by same
//elements, 2 nd row elements by square of each element and 3 rd row elements by cube of each element
//and so on.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void modifyRows(int *mat, int width)
{
    int rowId = threadIdx.x + blockIdx.x * blockDim.x;
    int element;

    for (int colId = 0; colId < width; colId++)
    {
        element = mat[rowId * width + colId];
        mat[rowId * width + colId] = powf(element, rowId + 1);
    }
}

int main()
{
    int matrix[100][100], linearMatrix[10000];
    int *dMatrix, numRows, numCols, matrixSize, index = 0;

    printf("Enter the number of rows and columns of the matrix: ");
    scanf("%d %d", &numRows, &numCols);
    matrixSize = numRows * numCols * sizeof(int);

    printf("Enter the matrix of size %dx%d:\n", numRows, numCols);

    for (int i = 0; i < numRows; i++)
    {
        for (int j = 0; j < numCols; j++)
        {
            scanf("%d", &matrix[i][j]);
            linearMatrix[index++] = matrix[i][j];
        }
    }

    hipMalloc((void **)&dMatrix, matrixSize);
    hipMemcpy(dMatrix, linearMatrix, matrixSize, hipMemcpyHostToDevice);

    modifyRows<<<1, numRows>>>(dMatrix, numCols);

    hipMemcpy(linearMatrix, dMatrix, matrixSize, hipMemcpyDeviceToHost);

    printf("Resultant Matrix:\n");
    index = 0;

    for (int i = 0; i < numRows; i++)
    {
        for (int j = 0; j < numCols; j++)
        {
            matrix[i][j] = linearMatrix[index++];
            printf("%d ", matrix[i][j]);
        }
        printf("\n");
    }

    hipFree(dMatrix);
    return 0;
}
