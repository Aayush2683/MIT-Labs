#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void spvm(float *data, int num_rows, int *col_index, int *row_ptr, float *x, float *y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows)
    {
        float result = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        for (int ele = row_start; ele < row_end; ++ele)
        {
            result += data[ele] * x[col_index[ele]];
        }
        y[row] = result;
    }
}

int main()
{
    int row, col, data_ind = 0;
    printf("Enter no of rows and columns of matrix A: ");
    scanf("%d %d", &row, &col);

    float *data = (float *)malloc(row * col * sizeof(float));
    float *x = (float *)malloc(row * sizeof(float));
    float *y = (float *)malloc(row * sizeof(float));
    int *row_ptr = (int *)malloc((row + 1) * sizeof(int));
    int *col_index = (int *)malloc(row * col * sizeof(int));
    row_ptr[0] = 0;

    printf("Enter sparse matrix A of size %dx%d:\n", row, col);
    for (int i = 0; i < row; ++i)
    {
        for (int j = 0; j < col; ++j)
        {
            scanf("%f", &data[i * col + j]);
            if (data[i * col + j] > 0)
            {
                col_index[data_ind] = j;
                data[data_ind++] = data[i * col + j];
            }
        }
        row_ptr[i + 1] = data_ind;
    }

    printf("Data: ");
    for (int i = 0; i < data_ind; ++i)
        printf("%f ", data[i]);
    printf("\nColumn_Index: ");
    for (int i = 0; i < data_ind; ++i)
        printf("%d ", col_index[i]);
    printf("\nRow_Ptr: ");
    for (int i = 0; i < row + 1; ++i)
        printf("%d ", row_ptr[i]);

    printf("\nEnter %d elements of x: ", col);
    for (int i = 0; i < row; ++i)
    {
        scanf("%f", &x[i]);
        printf("%f ", x[i]);
    }

    float *d_data, *d_x, *d_y;
    int *d_row_ptr, *d_col_index;

    hipMalloc((void **)&d_data, (data_ind + 1) * sizeof(float));
    hipMalloc((void **)&d_x, row * sizeof(float));
    hipMalloc((void **)&d_y, row * sizeof(float));
    hipMalloc((void **)&d_row_ptr, (row + 1) * sizeof(int));
    hipMalloc((void **)&d_col_index, data_ind * sizeof(int));

    hipMemcpy(d_data, data, (data_ind + 1) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, row * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, (row + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, col_index, data_ind * sizeof(int), hipMemcpyHostToDevice);

    spvm<<<ceil(row / 256.0), 256>>>(d_data, row, d_col_index, d_row_ptr, d_x, d_y);

    hipMemcpy(y, d_y, row * sizeof(float), hipMemcpyDeviceToHost);

    printf("\nFinal Vector:\n");
    for (int i = 0; i < row; ++i)
        printf("%f\n", y[i]);

    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_y);
    hipFree(d_x);

    free(data);
    free(x);
    free(y);
    free(row_ptr);
    free(col_index);

    return 0;
}
