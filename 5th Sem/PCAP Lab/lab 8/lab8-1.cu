//Write a program in CUDA to perform parallel Sparse Matrix - Vector multiplication using com-
//pressed sparse row (CSR) storage format. Represent the input sparse matrix in CSR format in the
//host code.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

// Define matrix dimensions (M and N)
#define M 4
#define N 4

__global__ void onesComplement(int *inMat, int *outMat, int rows, int cols)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r >= 0 && r < rows && c >= 0 && c < cols)
    {
        if (r > 0 && r < rows - 1 && c > 0 && c < cols - 1)
        {
            // Calculate the index for the current element
            int idx = r * cols + c;
            // Calculate the 1's complement of the element and store it in outMat
            int num = inMat[idx];
            int rev = 0;
            for (int i = 0; num > 0; i++)
            {
                rev *= 10;
                rev += 1 - num % 2;
                num = num / 2;
            }
            outMat[idx] = rev;
        }
        else
        {
            // Copy border elements as-is
            outMat[r * cols + c] = inMat[r * cols + c];
        }
    }
}

int main()
{
    int inMat[M][N];
    int outMat[M][N];

    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            inMat[i][j] = i * N + j;
        }
    }

    int *dInMat, *dOutMat; // Device pointers for matrices inMat and outMat

    hipMalloc((void **)&dInMat, M * N * sizeof(int));
    hipMalloc((void **)&dOutMat, M * N * sizeof(int));
    hipMemcpy(dInMat, inMat, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    onesComplement<<<numBlocks, threadsPerBlock>>>(dInMat, dOutMat, M, N);

    hipMemcpy(outMat, dOutMat, M * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Matrix inMat:\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", inMat[i][j]);
        }
        printf("\n");
    }

    printf("Matrix outMat (1's complement of non-border elements in binary):\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", outMat[i][j]);
        }
        printf("\n");
    }

    hipFree(dInMat);
    hipFree(dOutMat);

    return 0;
}
